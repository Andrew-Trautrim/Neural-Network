#include <hip/hip_runtime.h>
#include <functional>
#include <sstream>
#include <stdexcept>

#include "Matrix.cuh"
#include "MatrixExpr.cuh"
#include "MatrixKernals.cuh"

const int THREADS_PER_DIM = 16;

MatrixExpr::MatrixExpr(int m, int n, std::function<void (double*)> eval) : m(m), n(n), eval(eval)
{
}

MatrixExpr MatrixExpr::operator+(const Matrix& other) const
{
    std::function<void (double*)> expr = [this, other](double* accumulator)
    {
        eval(accumulator);

        // Set kernal parameters
        int blocks_y = (m + THREADS_PER_DIM - 1) / THREADS_PER_DIM;
        int blocks_x = (n + THREADS_PER_DIM - 1) / THREADS_PER_DIM;

        dim3 THREADS(THREADS_PER_DIM, THREADS_PER_DIM);
        dim3 BLOCKS(blocks_x, blocks_y);

        if (m == other.m && n == other.n)
        {
            MatrixKernals::add<<<BLOCKS,THREADS>>>(accumulator, other.data.get(), accumulator, m, n);
            hipDeviceSynchronize();
        }
        else if (m == other.m && other.n == 1)
        {
            MatrixKernals::add_broadcast_horizontal<<<BLOCKS,THREADS>>>(accumulator, other.data.get(), accumulator, m, n);
            hipDeviceSynchronize();
        }
        else if (n == other.n && other.m == 1)
        {
            MatrixKernals::add_broadcast_vertical<<<BLOCKS,THREADS>>>(accumulator, other.data.get(), accumulator, m, n);
            hipDeviceSynchronize();
        }
        else 
        {
            std::ostringstream err;
            err << "Invalid dimensions: cannot add "
                << m << "x" << n
                << " matrix to "
                << other.m << "x" << other.n
                << " matrix.";
            throw std::invalid_argument(err.str()); 
        }
    };

    return MatrixExpr(m, n, expr);
}

MatrixExpr MatrixExpr::operator-(const Matrix& other) const
{
    std::function<void (double*)> expr = [this, other](double* accumulator)
    {
        eval(accumulator);

        // Set kernal parameters
        int blocks_y = (m + THREADS_PER_DIM - 1) / THREADS_PER_DIM;
        int blocks_x = (n + THREADS_PER_DIM - 1) / THREADS_PER_DIM;

        dim3 THREADS(THREADS_PER_DIM, THREADS_PER_DIM);
        dim3 BLOCKS(blocks_x, blocks_y);

        if (m == other.m && n == other.n)
        {
            MatrixKernals::subtract<<<BLOCKS,THREADS>>>(accumulator, other.data.get(), accumulator, m, n);
            hipDeviceSynchronize();
        }
        else if (m == other.m && other.n == 1)
        {
            MatrixKernals::subtract_broadcast_horizontal<<<BLOCKS,THREADS>>>(accumulator, other.data.get(), accumulator, m, n);
            hipDeviceSynchronize();
        }
        else if (n == other.n && other.m == 1)
        {
            MatrixKernals::subtract_broadcast_vertical<<<BLOCKS,THREADS>>>(accumulator, other.data.get(), accumulator, m, n);
            hipDeviceSynchronize();
        }
        else 
        {
            std::ostringstream err;
            err << "Invalid dimensions: cannot subtract "
                << other.m << "x" << other.n
                << " matrix from "
                << m << "x" << n
                << " matrix.";
            throw std::invalid_argument(err.str()); 
        }
    };

    return MatrixExpr(m, n, expr);
}

MatrixExpr MatrixExpr::operator*(const Matrix& other) const
{
    if (m != other.m || n != other.n)
    {
        std::ostringstream err;
        err << "Invalid dimensions: cannot multiply "
            << m << "x" << n
            << " matrix with "
            << other.m << "x" << other.n
            << " matrix.";
        throw std::invalid_argument(err.str()); 
    }

    std::function<void (double*)> expr = [this, other](double* accumulator)
    {
        eval(accumulator);

        // Set kernal parameters
        int blocks_y = (m + THREADS_PER_DIM - 1) / THREADS_PER_DIM;
        int blocks_x = (n + THREADS_PER_DIM - 1) / THREADS_PER_DIM;

        dim3 THREADS(THREADS_PER_DIM, THREADS_PER_DIM);
        dim3 BLOCKS(blocks_x, blocks_y);

        // Execute kernal
        MatrixKernals::multiply<<<BLOCKS,THREADS>>>(accumulator, other.data.get(), accumulator, m, n);
        hipDeviceSynchronize();
    };

    return MatrixExpr(m, n, expr);
}

MatrixExpr MatrixExpr::operator/(const Matrix& other) const
{
    if (m != other.m || n != other.n)
    {
        std::ostringstream err;
        err << "Invalid dimensions: cannot multiply "
            << m << "x" << n
            << " matrix with "
            << other.m << "x" << other.n
            << " matrix.";
        throw std::invalid_argument(err.str()); 
    }

    std::function<void (double*)> expr = [this, other](double* accumulator)
    {
        eval(accumulator);

        // Set kernal parameters
        int blocks_y = (m + THREADS_PER_DIM - 1) / THREADS_PER_DIM;
        int blocks_x = (n + THREADS_PER_DIM - 1) / THREADS_PER_DIM;

        dim3 THREADS(THREADS_PER_DIM, THREADS_PER_DIM);
        dim3 BLOCKS(blocks_x, blocks_y);

        // Execute kernal
        MatrixKernals::divide<<<BLOCKS,THREADS>>>(accumulator, other.data.get(), accumulator, m, n);
        hipDeviceSynchronize();
    };

    return MatrixExpr(m, n, expr);
}


MatrixExpr MatrixExpr::operator+(double num) const
{
    std::function<void (double*)> expr = [this, num](double* accumulator)
    {
        eval(accumulator);

        // Set kernal parameters
        int blocks_y = (m + THREADS_PER_DIM - 1) / THREADS_PER_DIM;
        int blocks_x = (n + THREADS_PER_DIM - 1) / THREADS_PER_DIM;

        dim3 THREADS(THREADS_PER_DIM, THREADS_PER_DIM);
        dim3 BLOCKS(blocks_x, blocks_y);

        // Execute kernal
        MatrixKernals::add<<<BLOCKS,THREADS>>>(accumulator, num, accumulator, m, n);
        hipDeviceSynchronize();
    };

    return MatrixExpr(m, n, expr);
}

MatrixExpr MatrixExpr::operator*(double num) const
{
    std::function<void (double*)> expr = [this, num](double* accumulator)
    {
        eval(accumulator);

        // Set kernal parameters
        int blocks_y = (m + THREADS_PER_DIM - 1) / THREADS_PER_DIM;
        int blocks_x = (n + THREADS_PER_DIM - 1) / THREADS_PER_DIM;

        dim3 THREADS(THREADS_PER_DIM, THREADS_PER_DIM);
        dim3 BLOCKS(blocks_x, blocks_y);

        // Execute kernal
        MatrixKernals::multiply<<<BLOCKS,THREADS>>>(accumulator, num, accumulator, m, n);
        hipDeviceSynchronize();
    };

    return MatrixExpr(m, n, expr);
}

Matrix MatrixExpr::evaluate(const MatrixExpr& expr)
{
    Matrix result(expr.m, expr.n);
    expr.eval(result.data.get());

    return result;
}
