#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <stdexcept>

#include "MatrixCommon.cuh"
#include "MatrixKernals.cuh"

const int THREADS_PER_DIM = 16;

namespace MatrixCommon
{
    void add(double* a, double* b, double* c, int a_m, int a_n, int b_m, int b_n)
    {
        // Set kernal parameters
        int blocks_y = (a_m + THREADS_PER_DIM - 1) / THREADS_PER_DIM;
        int blocks_x = (a_n + THREADS_PER_DIM - 1) / THREADS_PER_DIM;

        dim3 THREADS(THREADS_PER_DIM, THREADS_PER_DIM);
        dim3 BLOCKS(blocks_x, blocks_y);

        if (a_m == b_m && a_n == b_n)
        {
            MatrixKernals::add<<<BLOCKS,THREADS>>>(a, b, c, a_m, a_n);
            hipDeviceSynchronize();
        }
        else if (a_m == b_m && b_n == 1)
        {
            MatrixKernals::add_broadcast_horizontal<<<BLOCKS,THREADS>>>(a, b, c, a_m, a_n);
            hipDeviceSynchronize();
        }
        else if (a_n == b_n && b_m == 1)
        {
            MatrixKernals::add_broadcast_vertical<<<BLOCKS,THREADS>>>(a, b, c, a_m, a_n);
            hipDeviceSynchronize();
        }
        else 
        {
            std::ostringstream err;
            err << "Invalid dimensions: cannot add "
                << a_m << "x" << a_n
                << " matrix and "
                << b_m << "x" << b_n
                << " matrix.";
            throw std::invalid_argument(err.str()); 
        }
    }

    void subtract(double* a, double* b, double* c, int a_m, int a_n, int b_m, int b_n)
    {
        // Set kernal parameters
        int blocks_y = (a_m + THREADS_PER_DIM - 1) / THREADS_PER_DIM;
        int blocks_x = (a_n + THREADS_PER_DIM - 1) / THREADS_PER_DIM;

        dim3 THREADS(THREADS_PER_DIM, THREADS_PER_DIM);
        dim3 BLOCKS(blocks_x, blocks_y);

        if (a_m == b_m && a_n == b_n)
        {
            MatrixKernals::subtract<<<BLOCKS,THREADS>>>(a, b, c, a_m, a_n);
            hipDeviceSynchronize();
        }
        else if (a_m == b_m && b_n == 1)
        {
            MatrixKernals::subtract_broadcast_horizontal<<<BLOCKS,THREADS>>>(a, b, c, a_m, a_n);
            hipDeviceSynchronize();
        }
        else if (a_n == b_n && b_m == 1)
        {
            MatrixKernals::subtract_broadcast_vertical<<<BLOCKS,THREADS>>>(a, b, c, a_m, a_n);
            hipDeviceSynchronize();
        }
        else 
        {
            std::ostringstream err;
            err << "Invalid dimensions: cannot subtract "
                << b_m << "x" << b_n
                << " matrix from "
                << a_m << "x" << a_n
                << " matrix.";
            throw std::invalid_argument(err.str()); 
        }
    }

    void multiply(double* a, double* b, double* c, int a_m, int a_n, int b_m, int b_n)
    {
        if (a_n != b_m)
        {
            std::ostringstream err;
            err << "Invalid dimensions: cannot multiply "
                << a_m << "x" << a_n
                << " matrix and "
                << b_m << "x" << b_n
                << " matrix.";
            throw std::invalid_argument(err.str()); 
        }

        // Set kernal parameters
        int blocks_y = (a_m + THREADS_PER_DIM - 1) / THREADS_PER_DIM;
        int blocks_x = (a_n + THREADS_PER_DIM - 1) / THREADS_PER_DIM;

        dim3 THREADS(THREADS_PER_DIM, THREADS_PER_DIM);
        dim3 BLOCKS(blocks_x, blocks_y);

        // Execute kernal
        MatrixKernals::multiply<<<BLOCKS,THREADS>>>(a, b, c, a_m, a_n, b_m, b_n);
        hipDeviceSynchronize();
    }

    void hadamardProduct(double* a, double* b, double* c, int a_m, int a_n, int b_m, int b_n)
    {
        if (a_m != b_m || a_n != b_n)
        {
            std::ostringstream err;
            err << "Invalid dimensions: cannot Hadamard product "
                << a_m << "x" << a_n
                << " matrix and "
                << b_m << "x" << b_n
                << " matrix.";
            throw std::invalid_argument(err.str()); 
        }

        // Set kernal parameters
        int blocks_y = (a_m + THREADS_PER_DIM - 1) / THREADS_PER_DIM;
        int blocks_x = (a_n + THREADS_PER_DIM - 1) / THREADS_PER_DIM;

        dim3 THREADS(THREADS_PER_DIM, THREADS_PER_DIM);
        dim3 BLOCKS(blocks_x, blocks_y);

        // Execute kernal
        MatrixKernals::hadamardProduct<<<BLOCKS,THREADS>>>(a, b, c, a_m, a_n);
        hipDeviceSynchronize();
    }

    void divide(double* a, double* b, double* c, int a_m, int a_n, int b_m, int b_n)
    {
        if (a_m != b_m || a_n != b_n)
        {
            std::ostringstream err;
            err << "Invalid dimensions: cannot divide "
                << a_m << "x" << a_n
                << " matrix by "
                << b_m << "x" << b_n
                << " matrix.";
            throw std::invalid_argument(err.str()); 
        }

        // Set kernal parameters
        int blocks_y = (a_m + THREADS_PER_DIM - 1) / THREADS_PER_DIM;
        int blocks_x = (a_n + THREADS_PER_DIM - 1) / THREADS_PER_DIM;

        dim3 THREADS(THREADS_PER_DIM, THREADS_PER_DIM);
        dim3 BLOCKS(blocks_x, blocks_y);

        // Execute kernal
        MatrixKernals::divide<<<BLOCKS,THREADS>>>(a, b, c, a_m, a_n);
        hipDeviceSynchronize();
    }
    
    void add(double* a, double num, double* b, int m, int n)
    {
        // Set kernal parameters
        int blocks_y = (m + THREADS_PER_DIM - 1) / THREADS_PER_DIM;
        int blocks_x = (n + THREADS_PER_DIM - 1) / THREADS_PER_DIM;

        dim3 THREADS(THREADS_PER_DIM, THREADS_PER_DIM);
        dim3 BLOCKS(blocks_x, blocks_y);

        // Execute kernal
        MatrixKernals::add<<<BLOCKS,THREADS>>>(a, num, b, m, n);
        hipDeviceSynchronize();
    }

    void subtract(double* a, double num, double* b, int m, int n)
    {
        // Set kernal parameters
        int blocks_y = (m + THREADS_PER_DIM - 1) / THREADS_PER_DIM;
        int blocks_x = (n + THREADS_PER_DIM - 1) / THREADS_PER_DIM;

        dim3 THREADS(THREADS_PER_DIM, THREADS_PER_DIM);
        dim3 BLOCKS(blocks_x, blocks_y);

        // Execute kernal
        MatrixKernals::subtract<<<BLOCKS,THREADS>>>(a, num, b, m, n);
        hipDeviceSynchronize();
    }

    void multiply(double* a, double num, double* b, int m, int n)
    {
        // Set kernal parameters
        int blocks_y = (m + THREADS_PER_DIM - 1) / THREADS_PER_DIM;
        int blocks_x = (n + THREADS_PER_DIM - 1) / THREADS_PER_DIM;

        dim3 THREADS(THREADS_PER_DIM, THREADS_PER_DIM);
        dim3 BLOCKS(blocks_x, blocks_y);

        // Execute kernal
        MatrixKernals::multiply<<<BLOCKS,THREADS>>>(a, num, b, m, n);
        hipDeviceSynchronize();
    }

    void divide(double* a, double num, double* b, int m, int n)
    {
        // Set kernal parameters
        int blocks_y = (m + THREADS_PER_DIM - 1) / THREADS_PER_DIM;
        int blocks_x = (n + THREADS_PER_DIM - 1) / THREADS_PER_DIM;

        dim3 THREADS(THREADS_PER_DIM, THREADS_PER_DIM);
        dim3 BLOCKS(blocks_x, blocks_y);

        // Execute kernal
        MatrixKernals::divide<<<BLOCKS,THREADS>>>(a, num, b, m, n);
        hipDeviceSynchronize();
    }

    void transpose(double* a, double* b, int m, int n)
    {
        // Set kernal parameters
        int blocks_y = (m + THREADS_PER_DIM - 1) / THREADS_PER_DIM;
        int blocks_x = (n + THREADS_PER_DIM - 1) / THREADS_PER_DIM;

        dim3 THREADS(THREADS_PER_DIM, THREADS_PER_DIM);
        dim3 BLOCKS(blocks_x, blocks_y);

        // Execute kernal
        MatrixKernals::transpose<<<BLOCKS,THREADS>>>(a, b, m, n);
        hipDeviceSynchronize();
    }

    void sum(double* a, double* b, int m, int n, int axis)
    {
        if (axis == 0)
        {
            // Set kernal parameters
            int blocks_x = (n + THREADS_PER_DIM - 1) / THREADS_PER_DIM;

            dim3 THREADS(THREADS_PER_DIM, THREADS_PER_DIM);
            dim3 BLOCKS(blocks_x);

            // Execute kernal
            MatrixKernals::sum_vertical<<<BLOCKS,THREADS>>>(a, b, m, n);
            hipDeviceSynchronize();

            return;
        }
        else if (axis == 1)
        {
            // Set kernal parameters
            int blocks_y = (m + THREADS_PER_DIM - 1) / THREADS_PER_DIM;

            dim3 THREADS(THREADS_PER_DIM, THREADS_PER_DIM);
            dim3 BLOCKS(blocks_y);

            // Execute kernal
            MatrixKernals::sum_horizontal<<<BLOCKS,THREADS>>>(a, b, m, n);
            hipDeviceSynchronize();

            return;
        }

        std::ostringstream err;
        err << "Unknown axis: " << axis << ".";
        throw std::invalid_argument(err.str()); 
    }

    void cross_entropy(double* a, double* b, double* c, int a_m, int a_n, int b_m, int b_n)
    {
        if (a_m != b_m || a_n != b_n)
        {
            std::ostringstream err;
            err << "Invalid dimensions: cannot compute cross entropy of "
                << a_m << "x" << a_n
                << " matrix and "
                << b_m << "x" << b_n
                << " matrix.";
            throw std::invalid_argument(err.str()); 
        }

        // Set kernal parameters
        int blocks_y = (a_m + THREADS_PER_DIM - 1) / THREADS_PER_DIM;
        int blocks_x = (a_n + THREADS_PER_DIM - 1) / THREADS_PER_DIM;

        dim3 THREADS(THREADS_PER_DIM, THREADS_PER_DIM);
        dim3 BLOCKS(blocks_x, blocks_y);

        // Execute kernal
        MatrixKernals::cross_entropy<<<BLOCKS,THREADS>>>(a, b, c, a_m, a_n);
        hipDeviceSynchronize();
    }

    void sigmoid(double* a, double* b, int m, int n)
    {
        // Set kernal parameters
        int blocks_y = (m + THREADS_PER_DIM - 1) / THREADS_PER_DIM;
        int blocks_x = (n + THREADS_PER_DIM - 1) / THREADS_PER_DIM;

        dim3 THREADS(THREADS_PER_DIM, THREADS_PER_DIM);
        dim3 BLOCKS(blocks_x, blocks_y);

        // Execute kernal
        MatrixKernals::sigmoid<<<BLOCKS,THREADS>>>(a, b, m, n);
        hipDeviceSynchronize();
    }

    void d_sigmoid(double* a, double* b, int m, int n)
    {
        // Set kernal parameters
        int blocks_y = (m + THREADS_PER_DIM - 1) / THREADS_PER_DIM;
        int blocks_x = (n + THREADS_PER_DIM - 1) / THREADS_PER_DIM;

        dim3 THREADS(THREADS_PER_DIM, THREADS_PER_DIM);
        dim3 BLOCKS(blocks_x, blocks_y);

        // Execute kernal
        MatrixKernals::d_sigmoid<<<BLOCKS,THREADS>>>(a, b, m, n);
        hipDeviceSynchronize();
    }

    void tanh(double* a, double* b, int m, int n)
    {
        // Set kernal parameters
        int blocks_y = (m + THREADS_PER_DIM - 1) / THREADS_PER_DIM;
        int blocks_x = (n + THREADS_PER_DIM - 1) / THREADS_PER_DIM;

        dim3 THREADS(THREADS_PER_DIM, THREADS_PER_DIM);
        dim3 BLOCKS(blocks_x, blocks_y);

        // Execute kernal
        MatrixKernals::tanh<<<BLOCKS,THREADS>>>(a, b, m, n);
        hipDeviceSynchronize();
    }

    void d_tanh(double* a, double* b, int m, int n)
    {
        // Set kernal parameters
        int blocks_y = (m + THREADS_PER_DIM - 1) / THREADS_PER_DIM;
        int blocks_x = (n + THREADS_PER_DIM - 1) / THREADS_PER_DIM;

        dim3 THREADS(THREADS_PER_DIM, THREADS_PER_DIM);
        dim3 BLOCKS(blocks_x, blocks_y);

        // Execute kernal
        MatrixKernals::d_tanh<<<BLOCKS,THREADS>>>(a, b, m, n);
        hipDeviceSynchronize();
    }

    void relu(double* a, double* b, int m, int n)
    {
        // Set kernal parameters
        int blocks_y = (m + THREADS_PER_DIM - 1) / THREADS_PER_DIM;
        int blocks_x = (n + THREADS_PER_DIM - 1) / THREADS_PER_DIM;

        dim3 THREADS(THREADS_PER_DIM, THREADS_PER_DIM);
        dim3 BLOCKS(blocks_x, blocks_y);

        // Execute kernal
        MatrixKernals::relu<<<BLOCKS,THREADS>>>(a, b, m, n);
        hipDeviceSynchronize();
    }

    void d_relu(double* a, double* b, int m, int n)
    {
        // Set kernal parameters
        int blocks_y = (m + THREADS_PER_DIM - 1) / THREADS_PER_DIM;
        int blocks_x = (n + THREADS_PER_DIM - 1) / THREADS_PER_DIM;

        dim3 THREADS(THREADS_PER_DIM, THREADS_PER_DIM);
        dim3 BLOCKS(blocks_x, blocks_y);

        // Execute kernal
        MatrixKernals::d_relu<<<BLOCKS,THREADS>>>(a, b, m, n);
        hipDeviceSynchronize();
    }

    void log(double* a, double* b, int m, int n)
    {
        // Set kernal parameters
        int blocks_y = (m + THREADS_PER_DIM - 1) / THREADS_PER_DIM;
        int blocks_x = (n + THREADS_PER_DIM - 1) / THREADS_PER_DIM;

        dim3 THREADS(THREADS_PER_DIM, THREADS_PER_DIM);
        dim3 BLOCKS(blocks_x, blocks_y);

        // Execute kernal
        MatrixKernals::log<<<BLOCKS,THREADS>>>(a, b, m, n);
        hipDeviceSynchronize();
    }
}